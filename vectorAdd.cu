#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// CUDA Kernel для вычисления среднего значения строки
__global__ void calculateRowAverages(const float* matrix, float* averages, int N) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < N) {
        float sum = 0.0f;
        for (int col = 0; col < N; ++col) {
            sum += matrix[row * N + col];
        }
        averages[row] = sum / N;

        // Отладочный вывод
        printf("Row %d: Sum = %f, Average = %f\n", row, sum, averages[row]);
    }
}

int main() {
    int N = 4; // Размер матрицы NxN
    size_t matrixSize = N * N * sizeof(float);
    size_t resultSize = N * sizeof(float);

    // Хостовая память для матрицы и результатов
    float* h_matrix = (float*)malloc(matrixSize);
    float* h_averages = (float*)malloc(resultSize);

    if (!h_matrix || !h_averages) {
        fprintf(stderr, "Failed to allocate host memory!\n");
        return EXIT_FAILURE;
    }

    // Чтение матрицы из файла
    FILE* inputFile = fopen("input_matrix.txt", "r");
    if (!inputFile) {
        fprintf(stderr, "Failed to open input file!\n");
        return EXIT_FAILURE;
    }

    for (int i = 0; i < N * N; ++i) {
        if (fscanf(inputFile, "%f", &h_matrix[i]) != 1) {
            fprintf(stderr, "Error reading matrix element at index %d\n", i);
            fclose(inputFile);
            return EXIT_FAILURE;
        }
    }
    fclose(inputFile);

    printf("Matrix on host:\n");
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            printf("%f ", h_matrix[i * N + j]);
        }
        printf("\n");
    }

    // Выделение памяти на устройстве
    float* d_matrix = nullptr;
    float* d_averages = nullptr;
    hipMalloc((void**)&d_matrix, matrixSize);
    hipMalloc((void**)&d_averages, resultSize);

    // Копирование матрицы на устройство
    hipMemcpy(d_matrix, h_matrix, matrixSize, hipMemcpyHostToDevice);

    // Конфигурация ядра
    int threadsPerBlock = N; // 4 потока для 4 строк
    int blocksPerGrid = 1;

    // Таймер CUDA
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    // Запуск CUDA ядра
    calculateRowAverages<<<blocksPerGrid, threadsPerBlock>>>(d_matrix, d_averages, N);

    // Проверка ошибок выполнения ядра
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA kernel failed: %s\n", hipGetErrorString(err));
        return EXIT_FAILURE;
    }
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Копирование результатов с устройства
    hipMemcpy(h_averages, d_averages, resultSize, hipMemcpyDeviceToHost);

    // Вывод результатов
    printf("Results from device:\n");
    for (int i = 0; i < N; ++i) {
        printf("Row %d Average: %f\n", i, h_averages[i]);
    }

    // Запись результатов в файл
    FILE* outputFile = fopen("result.txt", "w");
    if (!outputFile) {
        fprintf(stderr, "Failed to open result file for writing!\n");
        return EXIT_FAILURE;
    }
    for (int i = 0; i < N; ++i) {
        fprintf(outputFile, "Row %d Average: %f\n", i, h_averages[i]);
    }
    fprintf(outputFile, "Execution time: %f ms\n", milliseconds);
    fclose(outputFile);

    // Очистка памяти
    free(h_matrix);
    free(h_averages);
    hipFree(d_matrix);
    hipFree(d_averages);

    printf("Done\n");
    return 0;
}
